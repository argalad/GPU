#include <stdio.h>
#include "hipblas.h"
#include "matrix_mul.h"

// Host multiplication function
// Compute C = A * B
// hA is the height of A
// wA is the width of A
// wB is the width of B

extern "C"
void Mul(float* A, float* B, int hA, int wA, int wB, float* C)
{
	int size;
	hipblasHandle_t handle;
    const float alpha = 1.0f;
	const float beta  = 0.0f;

	// Load A and B to the device
	float* Ad;
	size = hA * wA * sizeof(float);
	hipMalloc((void**)&Ad, size);
	hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);
	float* Bd;
	size = wA * wB * sizeof(float);
	hipMalloc((void**)&Bd, size);
	hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);

	// Allocate C on the device
	float* Cd;
	size = hA * wB * sizeof(float);
	hipMalloc((void**)&Cd, size);

    hipblasCreate(&handle);
	// Compute the execution configuration
	hipblasSgemm( handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
		hA,				/* [m] */ 
		wB,				/* [n] */  
		wA,				/* [k] */ 
		&alpha,			/* alfa */ 
		A, wA,			/* A[m][k], num columnas (lda) */ 
		B, wB,			/* B[k][n], num columnas (ldb) */
		&beta,			/* beta */
		C, wB			/* C[m][n], num columnas (ldc) */
	);

	// Read C from the device
	hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);

	// Free device memory
	hipFree(Ad);
	hipFree(Bd);
	hipFree(Cd);
}

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "routinesGPU.h"

#define BLOCK_SIZE 16

__global__ void noiseReduction (uint8_t *im, float *NR, int height, int width)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x+2;
	int i = blockIdx.y * blockDim.y + threadIdx.y+2;
	int col = threadIdx.x+2;
	int row = threadIdx.y+2;

	__shared__ uint8_t subim[BLOCK_SIZE + 4][BLOCK_SIZE + 4];
	subim[row][col] = im[i*width + j];

	if (threadIdx.x == 0)
	{
		subim[threadIdx.y+2][threadIdx.x+1] = im[i*width + (j-1)];
		subim[threadIdx.y+2][threadIdx.x  ] = im[i*width + (j-2)];
	}
	else if (threadIdx.x == BLOCK_SIZE-1)
	{
		subim[threadIdx.y+2][threadIdx.x+3] = im[i*width + (j+1)];
		subim[threadIdx.y+2][threadIdx.x+4] = im[i*width + (j+2)];
	}

	if (threadIdx.y == 0)
	{
		subim[threadIdx.y+1][threadIdx.x+2] = im[(i-1)*width + j];
		subim[threadIdx.y  ][threadIdx.x+2] = im[(i-2)*width + j]; 
	}
	else if (threadIdx.y == BLOCK_SIZE-1)
	{
		subim[threadIdx.y+3][threadIdx.x+2] = im[(i+1)*width + j];
		subim[threadIdx.y+4][threadIdx.x+2] = im[(i+2)*width + j];
	}

	if (((i >= 2) && (i < height-2)) && ((j >= 2) && (j < width-2)))
	{
		NR[i*width + j] =
			(2.0*subim[row-2][col-2] +  4.0*subim[row-2][col-1] +  5.0*subim[row-2][col] +  4.0*subim[row-2][col+1] + 2.0*subim[row-2][col+2]
		   + 4.0*subim[row-1][col-2] +  9.0*subim[row-1][col-1] + 12.0*subim[row-1][col] +  9.0*subim[row-1][col+1] + 4.0*subim[row-1][col+2]
		   + 5.0*subim[row  ][col-2] + 12.0*subim[row  ][col-1] + 15.0*subim[row  ][col] + 12.0*subim[row  ][col+1] + 5.0*subim[row  ][col+2]
		   + 4.0*subim[row+1][col-2] +  9.0*subim[row+1][col-1] + 12.0*subim[row+1][col] +  9.0*subim[row+1][col+1] + 4.0*subim[row+1][col+2]
		   + 2.0*subim[row+2][col-2] +  4.0*subim[row+2][col-1] +  5.0*subim[row+2][col] +  4.0*subim[row+2][col+1] + 2.0*subim[row+2][col+2])
		   /159.0;
	}
}

__global__ void gradient (float *NR, float *G, float *phi, float *Gx, float *Gy, int height, int width)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	float PI = 3.141593;

	if(((i >= 2) && (i < height-2)) && ((j >= 2) && (j < width-2))) {
		Gx[i*width + j] = 
			 (1.0*NR[(i-2)*width + (j-2)] +  2.0*NR[(i-2)*width + (j-1)] +  (-2.0)*NR[(i-2)*width + (j+1)] + (-1.0)*NR[(i-2)*width + (j+2)]
			+ 4.0*NR[(i-1)*width + (j-2)] +  8.0*NR[(i-1)*width + (j-1)] +  (-8.0)*NR[(i-1)*width + (j+1)] + (-4.0)*NR[(i-1)*width + (j+2)]
			+ 6.0*NR[(i  )*width + (j-2)] + 12.0*NR[(i  )*width + (j-1)] + (-12.0)*NR[(i  )*width + (j+1)] + (-6.0)*NR[(i  )*width + (j+2)]
			+ 4.0*NR[(i+1)*width + (j-2)] +  8.0*NR[(i+1)*width + (j-1)] +  (-8.0)*NR[(i+1)*width + (j+1)] + (-4.0)*NR[(i+1)*width + (j+2)]
			+ 1.0*NR[(i+2)*width + (j-2)] +  2.0*NR[(i+2)*width + (j-1)] +  (-2.0)*NR[(i+2)*width + (j+1)] + (-1.0)*NR[(i+2)*width + (j+2)]);


		Gy[i*width + j] = 
			 ((-1.0)*NR[(i-2)*width + (j-2)] + (-4.0)*NR[(i-2)*width + (j-1)] +  (-6.0)*NR[(i-2)*width + (j)] + (-4.0)*NR[(i-2)*width + (j+1)] + (-1.0)*NR[(i-2)*width + (j+2)]
			+ (-2.0)*NR[(i-1)*width + (j-2)] + (-8.0)*NR[(i-1)*width + (j-1)] + (-12.0)*NR[(i-1)*width + (j)] + (-8.0)*NR[(i-1)*width + (j+1)] + (-2.0)*NR[(i-1)*width + (j+2)]
			+    2.0*NR[(i+1)*width + (j-2)] +    8.0*NR[(i+1)*width + (j-1)] +    12.0*NR[(i+1)*width + (j)] +    8.0*NR[(i+1)*width + (j+1)] +    2.0*NR[(i+1)*width + (j+2)]
			+    1.0*NR[(i+2)*width + (j-2)] +    4.0*NR[(i+2)*width + (j-1)] +     6.0*NR[(i+2)*width + (j)] +    4.0*NR[(i+2)*width + (j+1)] +    1.0*NR[(i+2)*width + (j+2)]);

		G[i*width + j] = sqrtf ((Gx[i*width + j]*Gx[i*width + j]) + (Gy[i*width + j]*Gy[i*width + j]));	//G = √Gx²+Gy²
		phi[i*width + j] = atan2f (fabs (Gy[i*width + j]), fabs (Gx[i*width + j]));

	if(fabs (phi[i*width + j]) <= PI/8 )
		phi[i*width + j] = 0;
	else if (fabs (phi[i*width + j]) <= 3*(PI/8))
		phi[i*width + j] = 45;
	else if (fabs (phi[i*width + j]) <= 5*(PI/8))
		phi[i*width + j] = 90;
	else if (fabs (phi[i*width + j]) <= 7*(PI/8))
		phi[i*width + j] = 135;
	else 
		phi[i*width + j] = 0;
	}
}

__global__ void pedge_calculation (float *G, float *phi, uint8_t *pedge, int height, int width)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if (((i >= 3) && (i < height-3)) && ((j >= 3) && (j < width-3))) {
		if (phi[i*width + j] == 0)
		{
			if (G[i*width + j] > G[i*width + j+1] && G[i*width + j] > G[i*width + j-1]) //edge is in N-S
				pedge[i*width + j] = 1;

		} else if (phi[i*width + j] == 45) {
			if (G[i*width + j] > G[(i+1)*width + j+1] && G[i*width + j] > G[(i-1)*width + j-1]) // edge is in NW-SE
				pedge[i*width + j] = 1;

		} else if (phi[i*width + j] == 90) {
			if (G[i*width + j] > G[(i+1)*width + j] && G[i*width + j] > G[(i-1)*width + j]) //edge is in E-W
				pedge[i*width + j] = 1;

		} else if (phi[i*width + j] == 135) {
			if (G[i*width + j] > G[(i+1)*width + j-1] && G[i*width + j] > G[(i-1)*width + j+1]) // edge is in NE-SW
				pedge[i*width + j] = 1;
		}
	}
}

__global__ void hysteresis_thresholding (uint8_t *image_out, float *G, uint8_t *pedge, float level, int height, int width)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int ii, jj;
	float lowthres = level/2;
	float hithres = 2*level;

	if (((i >= 3) && (i < height-3)) && ((j >= 3) && (j < width-3))) {
		if (G[i*width + j] > hithres && pedge[i*width + j])
			image_out[i*width + j] = 255;
		else if (pedge[i*width + j] && G[i*width + j] >= lowthres && G[i*width + j] < hithres)
			// check neighbours 3x3
			for (ii = -1; ii <= 1; ii++)
				for (jj = -1; jj <= 1; jj++)
					if (G[(i + ii)*width + j + jj] > hithres)
						image_out[i*width + j] = 255;
	}
}

__global__ void houghtransform_GPU (uint8_t *im, int width, int height, uint32_t *accumulators, 
	float *sin_table, float *cos_table, float hough_h)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int theta;

	float center_x = width/2.0; 
	float center_y = height/2.0;

	if (((i >= 0) && (i < height)) && ((j >= 0) && (j < width)))
	{
		if (im[(i*width) + j] > 250) // Pixel is edge  
		{  
			for (theta = 0; theta < 180; theta++)  
			{  
				float rho = (((float)j - center_x) * cos_table[theta]) + (((float)i - center_y) * sin_table[theta]);
				accumulators[(int)((round (rho + hough_h)*180.0)) + theta]++;
			} 
		} 
	}
}

__global__ void getlines_GPU (int threshold, uint32_t *accumulators, int accu_width, int accu_height, int width, int height, 
	float *sin_table, float *cos_table,
	int *x1_lines, int *y1_lines, int *x2_lines, int *y2_lines, int *lines)
{
	int theta = blockIdx.x * blockDim.x + threadIdx.x;
	int rho = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t max;

	if (((rho >= 0) && (rho < accu_height)) && ((theta >= 0) && (theta < accu_width)))
	{	
		if(accumulators[(rho*accu_width) + theta] >= threshold)  
		{ 
			//Is this point a local maxima (9x9)  
			max = accumulators[(rho*accu_width) + theta]; 
			for (int ii = -4; ii <= 4; ii++)  
			{  
				for (int jj = -4; jj <= 4; jj++)  
				{  
					if ((ii + rho >= 0 && ii + rho <accu_height) && (jj + theta >= 0 && jj + theta < accu_width))  
					{  
						if (accumulators[((rho + ii) * accu_width) + (theta + jj)] > max )
							max = accumulators[((rho + ii) * accu_width) + (theta + jj)];
					}  
				}  
			}  

			if(max == accumulators[(rho*accu_width) + theta]) //local maxima
			{
				int x1, y1, x2, y2;  
				x1 = y1 = x2 = y2 = 0; 

				if(theta >= 45 && theta <= 135)  
				{
					if (theta > 90) {
						//y = (r - x cos(t)) / sin(t)  
						x1 = width/2;  
						y1 = ((float)(rho - (accu_height/2)) - ((x1 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);
						x2 = width;  
						y2 = ((float)(rho - (accu_height/2)) - ((x2 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);  
					} else {
						//y = (r - x cos(t)) / sin(t)  
						x1 = 0;  
						y1 = ((float)(rho - (accu_height/2)) - ((x1 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);
						x2 = width * 2/5;  
						y2 = ((float)(rho - (accu_height/2)) - ((x2 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2); 
					}
				} else {
					//x = (r - y sin(t)) / cos(t);  
					y1 = 0;  
					x1 = ((float)(rho-(accu_height/2)) - ((y1 - (height/2) ) * sin_table[theta])) / cos_table[theta] + (width / 2);  
					y2 = height;  
					x2 = ((float)(rho-(accu_height/2)) - ((y2 - (height/2) ) * sin_table[theta])) / cos_table[theta] + (width / 2);  
				}
				x1_lines[*lines] = x1;
				y1_lines[*lines] = y1;
				x2_lines[*lines] = x2;
				y2_lines[*lines] = y2;
				(*lines)++;
			}
		}
	}
}

void line_asist_GPU (uint8_t *im, int height, int width,
	uint8_t *imEdge, float *NR, float *G, float *phi, float *Gx, float *Gy, uint8_t *pedge,
	float *sin_table, float *cos_table,
	uint32_t *accum, int accu_height, int accu_width,
	int *x1, int *x2, int *y1, int *y2, int *nlines)
{
	int img_size = height * width;
	uint8_t *im_gpu, *imEdge_gpu;
	float *NR_gpu;
	float *G_gpu;
	float *phi_gpu;
	float *Gx_gpu;
	float *Gy_gpu;
	uint8_t *pedge_gpu;
	uint32_t *accum_gpu;
	float *sin_table_gpu, *cos_table_gpu;
	int threshold;

	hipMalloc ((uint8_t**)&im_gpu, sizeof(uint8_t) * img_size);
	hipMalloc ((uint8_t**)&imEdge_gpu, sizeof (uint8_t) * img_size);
	hipMalloc ((float**)&NR_gpu, sizeof (float) * img_size);
	hipMalloc ((float**)&G_gpu, sizeof (float) * img_size);
	hipMalloc ((float**)&phi_gpu, sizeof (float) * img_size);
	hipMalloc ((float**)&Gx_gpu, sizeof (float) * img_size);
	hipMalloc ((float**)&Gy_gpu, sizeof (float) * img_size);
	hipMalloc ((uint8_t**)&pedge_gpu, sizeof (uint8_t) * img_size);
	hipMalloc ((uint32_t**)&accum_gpu, sizeof (uint32_t) * accu_width * accu_height);
	hipMalloc ((float**)&sin_table_gpu, sizeof (float) * 180);
	hipMalloc ((float**)&cos_table_gpu, sizeof (float) * 180);

	hipMemcpy (im_gpu, im, sizeof (uint8_t) * img_size, hipMemcpyHostToDevice);

	dim3 dimBlock (BLOCK_SIZE, BLOCK_SIZE);

	int dimblock1;
	if (height % BLOCK_SIZE == 0)
		dimblock1 = height/BLOCK_SIZE;
	else
		dimblock1 = height/BLOCK_SIZE+1;

	int dimblock2;
	if (width % BLOCK_SIZE == 0)
		dimblock2 = width/BLOCK_SIZE;
	else
		dimblock2 = width/BLOCK_SIZE+1;

	dim3 dimGrid(dimblock1, dimblock2);

	/* Canny */
	noiseReduction<<<dimGrid, dimBlock>>> (im_gpu, NR_gpu, height, width);
	hipDeviceSynchronize ();

	gradient<<<dimGrid, dimBlock>>> (NR_gpu, G_gpu, phi_gpu, Gx_gpu, Gy_gpu, height, width);
	hipDeviceSynchronize ();

	pedge_calculation<<<dimGrid, dimBlock>>> (G_gpu, phi_gpu, pedge_gpu, height, width);
	hipDeviceSynchronize ();

	hysteresis_thresholding<<<dimGrid, dimBlock>>> (imEdge_gpu, G_gpu, pedge_gpu, 1000.0f, height, width);
	hipDeviceSynchronize ();

	hipMemcpy (sin_table_gpu, sin_table, sizeof (float) * 180, hipMemcpyHostToDevice);
	hipMemcpy (cos_table_gpu, cos_table, sizeof (float) * 180, hipMemcpyHostToDevice);
	
	for(int k = 0; k < accu_width * accu_height; k++)
		accum[k] = 0;

	hipMemcpy (accum_gpu, accum, sizeof (uint32_t) * accu_width * accu_height, hipMemcpyHostToDevice);

	/* hough transform */
	float hough_h = ((sqrt(2.0) * (float)(height > width ? height : width)) / 2.0);	
	houghtransform_GPU<<<dimGrid, dimBlock>>> (imEdge_gpu, width, height, accum_gpu, sin_table_gpu, cos_table_gpu, hough_h);
	hipDeviceSynchronize ();

	if (width > height) threshold = width/6;
	else threshold = height/6;

	getlines_GPU<<<dimGrid, dimBlock>>> (threshold, accum_gpu, accu_width, accu_height, width, height,
		sin_table_gpu, cos_table_gpu,
		x1, y1, x2, y2, nlines);
	hipDeviceSynchronize ();

	hipFree (im_gpu);
	hipFree (imEdge_gpu);
	hipFree (NR_gpu);
	hipFree (G_gpu);
	hipFree (Gx_gpu);
	hipFree (Gy_gpu);
	hipFree (phi_gpu);
	hipFree (pedge_gpu);
	hipFree (accum_gpu);
	hipFree (sin_table_gpu);
	hipFree (cos_table_gpu);
}
